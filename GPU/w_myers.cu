#include "hip/hip_runtime.h"
/*
Authors: Alexsander Melo and Ygor Canalli
Date: November of 2014
Topicos Especiais em Programacao de Computadores = TEPC
Universidade Federal Rural do Rio de Janeiro - UFRRJ
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <string.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include "App.h"


extern "C" {
	#include "in.h"
}

typedef unsigned char boolean;
//typedef unsigned char bitvector_64;
//typedef unsigned short bitvector_64;
typedef unsigned int bitvector_64;
//typedef unsigned long bitvector_64;

//#define BITVECTOR_64_MAX UCHAR_MAX
//#define BITVECTOR_64_MAX USHRT_MAX
#define BITVECTOR_64_MAX UINT_MAX
//#define BITVECTOR_64_MAX ULONG_MAX

#define WORD_SIZE ((int) sizeof(bitvector_64) * 8)
#define ASCII pow(2, sizeof(char) * 8) /*max length of ASCII*/
//#define UTF8 (unsigned int) pow(2, sizeof(char) * 8) /*max length of UTF8*/

#define MASK(z) (bitvector_64) 1 << (z - 1)


#define GPU 0
#define MAX_THREADS 1024


#define CHECK_ERROR(call) do {\
   if( hipSuccess != call) {\
      fprintf(stderr,"\nCUDA ERROR:%s in file: %s in line: %d", hipGetErrorString(call),  __FILE__, __LINE__);\
         exit(0);\
} } while (0)



/*Functions Header*/
__device__ __host__ void printInfo(bitvector_64, bitvector_64, bitvector_64, bitvector_64, bitvector_64, bitvector_64, bitvector_64, long score);

__device__ __host__ char* strBin(bitvector_64);

void printoccurrences(boolean*, unsigned int);

boolean* myers(const char*, unsigned int, const char*, unsigned int, unsigned int, const unsigned int, const unsigned int, const unsigned int);

__global__ void myers_kernel(bitvector_64*, const unsigned int, const unsigned int, const bitvector_64, const bitvector_64, const unsigned int, const unsigned int, boolean*, const char*, const unsigned int, const unsigned int, const unsigned int);

void* preprocessingPEq(bitvector_64**, const char*, const unsigned int, const unsigned int);

__device__ long advancedBlock(bitvector_64*, const bitvector_64, const bitvector_64, unsigned int, bitvector_64*, bitvector_64*, bitvector_64*,	bitvector_64*, bitvector_64*, bitvector_64*, unsigned int);



int main(int argc, char** argv)
{

	string* strp; 
	string* strt;

	char *t = NULL, /*text*/
	     *p = NULL; /*pattern*/

	boolean* occurrence_array;


	unsigned int alphabet_lenght = ASCII;

	unsigned int m = 0;  /*length of pattern*/
	unsigned int n = 0; /*length of text*/

	unsigned int k = 0;
   unsigned int device = GPU;
   unsigned int threads_per_block = MAX_THREADS;
	
	short int c;
	
	while ((c = getopt (argc, argv, "P:T:p:t:a:k:d:b:")) != -1)
	{
		switch(c)
		{
			case 'P':
				strp = readTextFromFile(optarg);
         		p = strp->content;
				m = strp->len;
            break;
			case 'T':
				strt = readTextFromFile(optarg);
				t = strt->content;
				n = strt->len;
				break;
			case 'p':
				p = optarg;
				m = strlen(p);
				break;
			case 't':
				t = optarg;
				n = strlen(t);
				break;
			case 'a':
				alphabet_lenght = atoi(optarg);
				break;
			case 'k':
				k = atoi(optarg);
				break;
			case 'd':
				device = atoi(optarg);
				break;
			case 'b':
				threads_per_block = atoi(optarg);
				break;
			case '?':
				if ((optopt == 'P') || (optopt == 'T') || (optopt == 'p') || (optopt == 't') || (optopt == 'a') || (optopt == 'k') || (optopt == 'd') || (optopt == 'b'))
		       			fprintf(stderr, "Option -%c requires an argument.\n", optopt);
				else
		       			fprintf(stderr, "Unknown option `-%c'.\n", optopt);
				return 1;
		}
	}

	/*Aproximate string matching*/
	if(k > m)
		k = 0;

	Stopwatch sw;
	FREQUENCY(sw);
  	START_STOPWATCH(sw);

	occurrence_array = myers(t, n, p, m, k, alphabet_lenght, device, threads_per_block);

  	STOP_STOPWATCH(sw);

  	//printoccurrences(occurrence_array, n);
	printf("\nTotal time %lf (ms)\n", sw.mElapsedTime);

  
   free(occurrence_array);
   free(p);
   free(t);
   free(strt);
   free(strp);

	return 0;
}



boolean* myers(const char *t, unsigned int n, const char *p, unsigned int m, unsigned int k, const unsigned int alphabet_lenght, const unsigned int device, const unsigned int threads_per_block)
{
	unsigned int n_slices = floor((float) n / m);
	unsigned int slice_mask = 1;

	// find slice_base_mask = 2^k and greater than slice_base
	while (n_slices > slice_mask)
		slice_mask <<= 1;

    /*n_slices is the greather power of 2 less or equal to slice_base*/
	if ((n_slices > 1) && (n_slices < slice_mask))
		n_slices = (slice_mask >> 1);

	/*========================================================================================*/
	CHECK_ERROR(hipSetDevice(device));
	CHECK_ERROR(hipDeviceReset());
	/*========================================================================================*/


	/*========================================================================================*/
	bitvector_64 HMASK;
	bitvector_64 MMASK;

	bitvector_64 bits = m;

	MMASK = (bits == WORD_SIZE) ? BITVECTOR_64_MAX : pow(2, bits) - 1;
	HMASK = (bits == WORD_SIZE) ? MASK(WORD_SIZE) : MASK(bits);
	/*========================================================================================*/


	/*========================================================================================*/
	bitvector_64 *PEq;
	bitvector_64 *d_PEq;

	PEq = (bitvector_64*) malloc (sizeof(bitvector_64) * alphabet_lenght);
	preprocessingPEq(&PEq, p, m, alphabet_lenght);

	CHECK_ERROR(hipMalloc(&d_PEq, alphabet_lenght * sizeof(bitvector_64)));

	/*Copy to device*/
	CHECK_ERROR(hipMemcpy(d_PEq, PEq, alphabet_lenght * sizeof(bitvector_64), hipMemcpyHostToDevice));	
	
	free(PEq);
	/*========================================================================================*/


	/*========================================================================================*/
	boolean *occurrence_array;
	boolean *d_occurrence_array;

	occurrence_array = (boolean*) malloc (sizeof(boolean) * n);
	CHECK_ERROR(hipMalloc((void**) &d_occurrence_array, n * sizeof(boolean)));
	CHECK_ERROR(hipMemset(d_occurrence_array, 0,  n * sizeof(boolean)));
	/*========================================================================================*/



	/*========================================================================================*/
	char* d_t;
	CHECK_ERROR(hipMalloc((void**) &d_t, (n * sizeof(char)) + 1));
	/*Copy to device*/
	CHECK_ERROR(hipMemcpy(d_t, t, (n * sizeof(char)), hipMemcpyHostToDevice));
	/*========================================================================================*/


	/*========================================================================================*/
	/*size_t _free = 0,
            total = 0;*/

   /*CHECK_ERROR(hipMemGetInfo(&_free, &total));
   printf("Memoria livre: %f MB", ((float)_free / 1024 / 1024));
   printf("Memoria total: %f MB", ((float)total / 1024 / 1024));*/
	/*========================================================================================*/


	/*========================================================================================*/
	unsigned int N_CUDA_THREADS = (n_slices > threads_per_block) ? threads_per_block : n_slices;
	unsigned int N_CUDA_BLOCKS = ceil((float) n_slices / threads_per_block);

	printf("\nSlices: %d, Threads: %d - Blocks: %d", n_slices, N_CUDA_THREADS, N_CUDA_BLOCKS);
	dim3 threadsPerBlock(N_CUDA_THREADS, 1);
  	dim3 numBlocks(N_CUDA_BLOCKS, 1);

	unsigned int slice_base = floor((float) n / n_slices);
	unsigned int remaining = n % n_slices;

   printf("\nLenght of text: %d - Slice base: %d - Remaining: %d", n, slice_base, remaining);

	Stopwatch sw;
	FREQUENCY(sw);
  	START_STOPWATCH(sw);

  	/*call myers kernel CUDA*/
	myers_kernel<<<numBlocks, threadsPerBlock>>>(d_PEq, slice_base, remaining, MMASK, HMASK, m, k, d_occurrence_array, d_t, n, n_slices, alphabet_lenght);
	CHECK_ERROR(hipDeviceSynchronize());

  	STOP_STOPWATCH(sw);
	printf("\nKernel time %lf (ms)", sw.mElapsedTime);
	/*========================================================================================*/

	

	/*========================================================================================*/
	/*Copy device to host*/
	CHECK_ERROR(hipMemcpy(occurrence_array, d_occurrence_array, n * sizeof(boolean), hipMemcpyDeviceToHost));
	/*========================================================================================*/

	

	/*========================================================================================*/
	/*free GPU memory*/
	CHECK_ERROR(hipFree(d_PEq));
	CHECK_ERROR(hipFree(d_occurrence_array));
	CHECK_ERROR(hipFree(d_t));
	/*========================================================================================*/


	return occurrence_array;
}



__global__ void myers_kernel(bitvector_64* PEq,
                      const unsigned int slice_base,
                      const unsigned int remaining,
							 const bitvector_64 MMASK,
							 const bitvector_64 HMASK,
							 const unsigned int m,
							 const unsigned int k,
							 boolean* occurrence_array,
							 const char* t,
							 const unsigned int n,
							 const unsigned int n_slices,
							 const unsigned int alphabet_lenght)
{

	unsigned int thread_reference = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int cooperation_remaining = (thread_reference < remaining);
   	unsigned int offset = (thread_reference * (slice_base + (remaining > 0))) - ((thread_reference - remaining)*(1-cooperation_remaining));
   	unsigned int overlapping_length = (thread_reference < (n_slices - 1))*(m - 1);
   	unsigned int length = slice_base + overlapping_length + cooperation_remaining;

	unsigned int position = 0;

    boolean signal = 0;
	unsigned int i = 0;

    bitvector_64 VP, VN, VX, HX, HP, HN;
	long score = m;

	/*set initial values*/
	VP = MMASK;
	VN = 0;

	for(i = 0; i < length; i++)
	{
		position = offset + i;
		
		score += advancedBlock(PEq, MMASK, HMASK, (unsigned int) t[position], &VP, &VN, &VX, &HX, &HP, &HN, alphabet_lenght);
		
		signal = score <= k;

		// We not found a better solution.
		// The time is not worst, but making more store operations we
		// spend a little bit more time than divergent memory acess
		signal && (occurrence_array[position] = signal);
	}	
}



void* preprocessingPEq(bitvector_64** PEq, const char* p, const unsigned int m, const unsigned int alphabet_lenght)
{
	unsigned int i = 0;

	for (i = 0; i < alphabet_lenght; i++)
		(*PEq)[i] = 0;
	

	for (i = 0; i < m; i++)
	{
		(*PEq)[((unsigned int) p[i])] |= MASK(i+1);
	}

	return 0;	
}


__device__ long advancedBlock(bitvector_64* PEq,
	 						const bitvector_64 MMASK,
	 						const bitvector_64 HMASK,
							unsigned int character_offset,
							bitvector_64* VP,
							bitvector_64* VN,
							bitvector_64* VX,
							bitvector_64* HX,   
							bitvector_64* HP,
							bitvector_64* HN,
							unsigned int alphabet_lenght)
{
	long score = 0;
	bitvector_64 EQ = 0;

	EQ = PEq[character_offset];

	(*VX) = EQ | *VN;
	(*VX) &= MMASK;

	(*HX) = (((EQ & *VP) + *VP) ^ *VP) | EQ;
	(*HX) &= MMASK;
	
	(*HP) = *VN | ~(*VP | *HX);
	(*HP) &= MMASK;

	(*HN) = *VP & *HX;
	(*HN) &= MMASK;

	score = ((*HP & HMASK) != 0) - ((*HN & HMASK) != 0);
		
	(*HP) = (*HP << 1);
	(*HP) &= MMASK;

	(*HN) <<= 1;
	(*HN) &= MMASK;

	(*VP) = *HN | ~(*HP | *VX);
	(*VP) &= MMASK;

	(*VN) = *HP & *VX;
	(*VN) &= MMASK;

	return score;
}



void printoccurrences(boolean* occurrence_array, unsigned int n)
{

	unsigned int i = 0;
	unsigned int total = 0;

	for (i = 0; i < n; i++)
	{
        if (occurrence_array[i])
		{
			printf("Occurrence at position: %d of text\n", i + 1);
			total++;
		}
	}

	printf("\nTotal of occurrences: %d\n", total);
}



__device__ __host__ char* strBin(bitvector_64 n)
{
    //unsigned short int n_bits = 0;
    char* str;
    int i = 0;

    /*Suppose n_bits <= WORD_SIZE*/
    //n_bits = (n > 1) ? floor(log(n) / log(2) + 0.5) : (n == 1);
    str = (char*) malloc (WORD_SIZE * sizeof(char) + 1);

    bitvector_64 numerator = n;

    for(i = WORD_SIZE - 1; i >= 0; i--)
    {    
        str[i] = 48 + (numerator % 2);        
        numerator >>= 1;
    }

    str[WORD_SIZE] = 0;

    return str;
}


__device__ __host__ void printInfo(bitvector_64 EQ, bitvector_64 VX, bitvector_64 HX, bitvector_64 HP, bitvector_64 HN, bitvector_64 VP, bitvector_64 VN, long score)
{
	printf("\nScore: %ld", score);
	printf("\nEQ: %s  ::  %u", strBin(EQ), EQ);
	printf("\nVX: %s  ::  %u", strBin(VX), VX);
	printf("\nHX: %s  ::  %u", strBin(HX), HX);
	printf("\nHP: %s  ::  %u", strBin(HP), HP);
	printf("\nVP: %s  ::  %u", strBin(VP), VP);
	printf("\nVN: %s  ::  %u", strBin(VN), VN);
	printf("\nHN: %s  ::  %u\n", strBin(HN), HN);
}
